#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "utilities.h"
using std::cout;
using std::endl;

template <typename T>
__global__ void sumArrays_gpu(T* a, T* b, T* ans, unsigned int size){
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < size)
        ans[tid] = a[tid] + b[tid];
}

template <typename T>
__global__ void initArr_gpu(T *data, unsigned int size){
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < size){
        data[tid] = 1;
    }
}

int main(int argc, char** argv){
    // create arr on host
    constexpr int size = 1 << 14; // 2 ^ 14 = 16KB

    // config for cuda
    initDevice(0);
    int blockSize_x = 1024; 
    if(argc > 1) blockSize_x = atoi(argv[1]);
    dim3 blockSize(blockSize_x, 1);
    dim3 gridSize((size + blockSize.x - 1) / blockSize.x, 1);


    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2);
    hipEventQuery(start2);
    

    for(int i = 0; i < 10; ++i){

        // without UVA
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        hipEventQuery(start);

        int *h_a = new int[size];
        int *h_b = new int[size];
        int *h_ans = new int[size];
        initArr(h_a, size);
        initArr(h_b, size);
        
        int *d_a=nullptr, *d_b=nullptr, *d_ans=nullptr;
        hipMalloc((void**)(&d_a), size*sizeof(int));
        hipMalloc((void**)(&d_b), size*sizeof(int));
        hipMalloc((void**)(&d_ans), size*sizeof(int));
        hipMemcpy(d_a, h_a, size*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, size*sizeof(int), hipMemcpyHostToDevice);
        
        sumArrays_gpu<int> <<<gridSize, blockSize>>> (d_a, d_b, d_ans, size);

        hipMemcpy(h_ans, d_ans, size*sizeof(int), hipMemcpyDeviceToHost);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float time_without_UVA = 0;
        hipEventElapsedTime(&time_without_UVA, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);

        cout<<"time without UVA:"<<time_without_UVA<<endl;
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_ans);
        delete[] h_a;
        delete[] h_b;
        delete[] h_ans;

        hipDeviceSynchronize();

        // with UVA
        hipEvent_t start2, stop2;
        hipEventCreate(&start2);
        hipEventCreate(&stop2);
        hipEventRecord(start2);
        hipEventQuery(start2);

        int *a = nullptr, *b = nullptr, *ans = nullptr;
        // hipMalloc((void**)(&a), size*sizeof(int));
        // hipMalloc((void**)(&b), size*sizeof(int));
        // hipMalloc((void**)(&ans), size*sizeof(int));
        CHECK(hipMallocManaged((void**)(&a), size*sizeof(int), hipMemAttachGlobal)); // a on Device
        // hipMemAdvise(a, size*sizeof(int), hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
        // CHECK(hipMemPrefetchAsync(a, size, hipCpuDeviceId));
        CHECK(hipMallocManaged((void**)(&b), size*sizeof(int), hipMemAttachGlobal)); // b on Device
        // hipMemAdvise(b, size*sizeof(int), hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
        // CHECK(hipMemPrefetchAsync(b, size, hipCpuDeviceId));
        CHECK(hipMallocManaged((void**)(&ans), size*sizeof(int), hipMemAttachGlobal)); // ans on Device
        // hipMemAdvise(ans, size*sizeof(int), hipMemAdviseSetPreferredLocation, 0);
        // initArr_gpu<int> <<<gridSize, blockSize>>> (a, size);
        // initArr_gpu<int> <<<gridSize, blockSize>>> (b, size);
        initArr(a, size);
        // CHECK(hipMemPrefetchAsync(a, size, 0));
        initArr(b, size);
        // CHECK(hipMemPrefetchAsync(b, size, 0));
        // hipDeviceSynchronize();
        sumArrays_gpu<int> <<<gridSize, blockSize>>> (a, b, ans, size); 

        // CHECK(hipMemPrefetchAsync(ans, size, hipCpuDeviceId)); // ans copy to Host

        hipEventRecord(stop2);
        hipEventSynchronize(stop2);
        float time_with_UVA = 0;
        hipEventElapsedTime(&time_with_UVA, start2, stop2);
        hipEventDestroy(start2);
        hipEventDestroy(stop2);

        cout<<"time with UVA:"<<time_with_UVA<<endl;
        hipFree(a);
        hipFree(b);
        hipFree(ans);

        hipDeviceSynchronize();
    }

    return 0;
}